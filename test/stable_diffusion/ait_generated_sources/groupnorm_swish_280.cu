#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include "cutlass/arch/memory_sm80.h"
#include "cutlass/cutlass.h"
#include "cutlass/fast_math.h"
#include "logging.h"




namespace {

//  Copyright (c) Meta Platforms, Inc. and affiliates.
//
//  Licensed under the Apache License, Version 2.0 (the "License");
//  you may not use this file except in compliance with the License.
//  You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
//
#ifndef GROUPNORM_KERNEL_CUH
#define GROUPNORM_KERNEL_CUH

#define FINAL_MASK 0xffffffff

#ifndef GROUP_NORM_CUDA_CHECK
#define GROUP_NORM_CUDA_CHECK(expr)                                       \
  do {                                                                    \
    hipError_t status = (expr);                                          \
    if (status != hipSuccess) {                                          \
      std::cerr << "CUDA error: " << hipGetErrorString(status) << " at " \
                << __FILE__ << ": " << __LINE__ << std::endl;             \
      return status;                                                      \
    }                                                                     \
  } while (0)
#endif

#ifndef GROUP_NORM_CUDA_CHECK_LAUNCH
#define GROUP_NORM_CUDA_CHECK_LAUNCH() GROUP_NORM_CUDA_CHECK(hipGetLastError())
#endif

__inline__ __device__ float sigmoid(float val) {
  return (cutlass::fast_tanh(val * 0.5f) + 1.0f) * 0.5f;
}

////////////////////////////////////////////////////////////////////////////////
// The Groupnorm implementation below is based on OneFlow's Layernorm
// implementation at:
// https://github.com/Oneflow-Inc/oneflow/blob/master/oneflow/core/cuda/layer_norm.cuh

/*
Copyright 2020 The OneFlow Authors. All rights reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#define __AIT_GN_USE_FAST_MATH 1
template <typename T>
__forceinline__ __device__ T Div(T a, T b);

template <>
__forceinline__ __device__ float Div<float>(float a, float b) {
#ifdef __AIT_GN_USE_FAST_MATH
  return __fdividef(a, b);
#else
  return a / b;
#endif
}

template <>
__forceinline__ __device__ half Div<half>(half a, half b) {
  return __hdiv(a, b);
}

template <typename T>
__forceinline__ __device__ T Rsqrt(T x);

template <>
__forceinline__ __device__ float Rsqrt<float>(float x) {
#ifdef __AIT_GN_USE_FAST_MATH
  return __frsqrt_rn(x);
#else
  return rsqrt(x);
#endif
}

template <>
__forceinline__ __device__ half Rsqrt<half>(half x) {
  return hrsqrt(x);
}

#undef __AIT_GN_USE_FAST_MATH

template <typename T>
inline __device__ void WelfordCombine(T val, T* mean, T* m2, int* count) {
  // Use Welford Online algorithem to compute mean and variance
  // For more details you can refer to:
  // https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Welford's_online_algorithm
  *count += 1;
  T delta1 = val - *mean;
  *mean += Div(delta1, static_cast<T>(*count));
  T delta2 = val - *mean;
  *m2 += delta1 * delta2;
}

template <typename T>
inline __device__ void WelfordCombine(
    T b_mean,
    T b_m2,
    int b_count,
    T* mean,
    T* m2,
    int* count) {
  if (b_count == 0) {
    return;
  }
  int new_count = *count + b_count;
  T nb_over_n = Div((T)b_count, (T)new_count);
  T delta = b_mean - *mean;
  *mean += delta * nb_over_n;
  *m2 += b_m2 + delta * delta * (T)(*count) * (T)(nb_over_n);
  *count = new_count;
}

constexpr int kWarpSize = 32;

template <typename T, int thread_group_width = kWarpSize>
__inline__ __device__ void WelfordWarpReduce(
    T thread_mean,
    T thread_m2,
    int thread_count,
    T* mean,
    T* m2,
    int* count) {
  *mean = thread_mean;
  *m2 = thread_m2;
  *count = thread_count;
  for (int mask = thread_group_width / 2; mask > 0; mask /= 2) {
    T b_mean = __shfl_down_sync(0xffffffff, *mean, mask, thread_group_width);
    T b_m2 = __shfl_down_sync(0xffffffff, *m2, mask, thread_group_width);
    int b_count =
        __shfl_down_sync(0xffffffff, *count, mask, thread_group_width);
    WelfordCombine(b_mean, b_m2, b_count, mean, m2, count);
  }
}

template <typename T>
__inline__ __device__ void WelfordBlockAllReduce(
    T thread_mean,
    T thread_m2,
    int thread_count,
    T* result_mean,
    T* result_m2,
    int* result_count) {
  __shared__ T mean_shared[kWarpSize];
  __shared__ T m2_shared[kWarpSize];
  __shared__ int count_shared[kWarpSize];
  __shared__ T mean_result_broadcast;
  __shared__ T m2_result_broadcast;
  __shared__ int count_result_broadcast;
  const int lid = threadIdx.x % kWarpSize;
  const int wid = threadIdx.x / kWarpSize;
  T warp_mean = 0;
  T warp_m2 = 0;
  int warp_count = 0;
  WelfordWarpReduce(
      thread_mean, thread_m2, thread_count, &warp_mean, &warp_m2, &warp_count);
  __syncthreads();
  if (lid == 0) {
    mean_shared[wid] = warp_mean;
    m2_shared[wid] = warp_m2;
    count_shared[wid] = warp_count;
  }
  __syncthreads();
  if (wid == 0) {
    if (threadIdx.x < blockDim.x / kWarpSize) {
      warp_mean = mean_shared[lid];
      warp_m2 = m2_shared[lid];
      warp_count = count_shared[lid];
    } else {
      warp_mean = static_cast<T>(0);
      warp_m2 = static_cast<T>(0);
      warp_count = static_cast<T>(0);
    }
    __syncwarp();
    T block_mean = 0;
    T block_m2 = 0;
    int block_count = 0;
    WelfordWarpReduce(
        warp_mean, warp_m2, warp_count, &block_mean, &block_m2, &block_count);
    if (lid == 0) {
      mean_result_broadcast = block_mean;
      m2_result_broadcast = block_m2;
      count_result_broadcast = block_count;
    }
  }
  __syncthreads();
  *result_mean = mean_result_broadcast;
  *result_m2 = m2_result_broadcast;
  *result_count = count_result_broadcast;
}

template <typename T, typename ComputeType, bool FuseSwish>
__global__ void groupnorm_welford_fp16(
    T* output,
    T* input,
    T* gamma,
    T* beta,
    const float eps,
    const int64_t elems_per_block,
    const int64_t elems_per_group_channel,
    const int64_t batch_stride,
    const int64_t group_stride,
    const int64_t num_rows,
    const int64_t row_stride) {
  // all the numbers and strides are counted with respect to type T
  constexpr int vec_size = sizeof(T) / sizeof(half);

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int gid = blockIdx.y; // index of group
  const int64_t batch_offset = bid * batch_stride;
  const int64_t group_offset = gid * group_stride;
  const int64_t offset = batch_offset + group_offset;

  // the first input of this thread
  const T* t_input = input + offset;

  ComputeType thread_mean = ComputeType(0.0);
  ComputeType thread_m2 = ComputeType(0.0);
  int thread_count = 0;
#pragma unroll
  for (int row_id = tid; row_id < num_rows; row_id += blockDim.x) {
#pragma unroll
    for (int i = 0; i < elems_per_group_channel; i++) {
      const T* local_input = t_input + i + row_id * row_stride;
      const half* half_ptr = reinterpret_cast<const half*>(local_input);
#pragma unroll
      for (int j = 0; j < vec_size; ++j) {
        WelfordCombine(
            __half2float(half_ptr[j]), &thread_mean, &thread_m2, &thread_count);
      }
    }
  }
  ComputeType row_mean = (ComputeType)(0.0f);
  ComputeType row_m2 = (ComputeType)(0.0f);
  int row_count = 0;
  if (blockDim.x <= 32) {
    WelfordWarpReduce(
        thread_mean, thread_m2, thread_count, &row_mean, &row_m2, &row_count);
  } else {
    WelfordBlockAllReduce<ComputeType>(
        thread_mean, thread_m2, thread_count, &row_mean, &row_m2, &row_count);
  }
  ComputeType row_variance = Div(row_m2, static_cast<ComputeType>(row_count));
  ComputeType row_inv_var = Rsqrt(row_variance + static_cast<ComputeType>(eps));

  float local_row_mean;
  if (std::is_same<ComputeType, half>::value) {
    local_row_mean = __half2float(row_mean);
  } else if (std::is_same<ComputeType, float>::value) {
    local_row_mean = row_mean;
  }
  float local_row_inv_var;
  if (std::is_same<ComputeType, half>::value) {
    local_row_inv_var = __half2float(row_inv_var);
  } else if (std::is_same<ComputeType, float>::value) {
    local_row_inv_var = row_inv_var;
  }

  const T* t_gamma = gamma + group_offset;
  const T* t_beta = beta + group_offset;
  // the first input of this thread
  T* t_output = output + offset;
#pragma unroll
  for (int row_id = tid; row_id < num_rows; row_id += blockDim.x) {
#pragma unroll
    for (int i = 0; i < elems_per_group_channel; i++) {
      const T* local_input = t_input + i + row_id * row_stride;
      const half* input_half_ptr = reinterpret_cast<const half*>(local_input);

      T* local_output = t_output + i + row_id * row_stride;
      T tmp_output;
      half* output_half_ptr = reinterpret_cast<half*>(&tmp_output);

      const T* local_gamma = t_gamma + i;
      const T* local_beta = t_beta + i;
      const half* gamma_half_ptr = reinterpret_cast<const half*>(local_gamma);
      const half* beta_half_ptr = reinterpret_cast<const half*>(local_beta);

#pragma unroll
      for (int j = 0; j < vec_size; ++j) {
        float local_val = __half2float(input_half_ptr[j]);
        float local_gamma = __half2float(gamma_half_ptr[j]);
        float local_beta = __half2float(beta_half_ptr[j]);
        float out_val = (local_val - local_row_mean) * local_row_inv_var;
        out_val = out_val * local_gamma + local_beta;
        out_val = FuseSwish ? out_val * sigmoid(out_val) : out_val;
        output_half_ptr[j] = __float2half_rn(out_val);
      }
      *local_output = tmp_output;
    }
  }
}

// End the Groupnorm implementation that is based on from OneFlow's Layernorm
////////////////////////////////////////////////////////////////////////////////

template <typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const {
    return a + b;
  }
};

template <template <typename> class ReductionOp, typename T, int block_size>
__inline__ __device__ T BlockAllReduce(T val) {
  typedef hipcub::BlockReduce<T, block_size> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  __shared__ T result_broadcast;
  T result = BlockReduce(temp_storage).Reduce(val, ReductionOp<T>());
  if (threadIdx.x == 0) {
    result_broadcast = result;
  }
  __syncthreads();
  return result_broadcast;
}

template <
    bool FuseSwish,
    int H,
    int W,
    int C,
    int C_G,
    int ILP = 8,
    int BANK_CONFLICT = 0,
    int NUM_THREADS = 1024>
__global__ __launch_bounds__(NUM_THREADS) void group_norm_smem(
    const half* X,
    half* Y,
    half* gamma,
    half* beta,
    int N,
    float epsilon) {
  constexpr int C_G_2 = C_G / 2;
  constexpr int C_G_stride = C_G_2 + BANK_CONFLICT;
  extern __shared__ int svals_[];
  auto* svals = reinterpret_cast<__half2*>(&svals_[0]);

  int32_t g = blockIdx.x;
  int32_t start_c = g * C_G;
  int32_t n = blockIdx.y;

  // X: [N, H, W, C]
  int32_t strides[4] = {H * W * C, W * C, C, 1};
  for (int32_t load_idx = threadIdx.x; load_idx < H / ILP * W * C_G_2;
       load_idx += blockDim.x) {
    auto c_g_2 = load_idx % C_G_2;
    auto w = (load_idx / C_G_2) % W;
    auto h_ilp = ((load_idx / C_G_2) / W);

#pragma unroll ILP
    for (auto ii = 0; ii < ILP; ++ii) {
      const __half2* src = reinterpret_cast<const __half2*>(
          &(X[n * strides[0] + (h_ilp * ILP + ii) * strides[1] +
              w * strides[2] + (start_c + c_g_2 * 2)]));
      __half2* dst =
          &svals[(h_ilp * ILP + ii) * W * C_G_stride + w * C_G_stride + c_g_2];
      cutlass::arch::cp_async_zfill<sizeof(__half2)>(dst, src, true);
    }
  }
  cutlass::arch::cp_async_wait<0>();

  float thread_sum = 0;
  for (int32_t load_idx = threadIdx.x; load_idx < H / ILP * W * C_G_2;
       load_idx += blockDim.x) {
    auto c_g_2 = load_idx % C_G_2;
    auto w = (load_idx / C_G_2) % W;
    auto h_ilp = ((load_idx / C_G_2) / W);
#pragma unroll ILP
    for (auto ii = 0; ii < ILP; ++ii) {
      half2 valh =
          svals[(h_ilp * ILP + ii) * W * C_G_stride + w * C_G_stride + c_g_2];
      float2 val = __half22float2(valh);
      thread_sum += val.x + val.y;
    }
  }
  const float block_mean =
      BlockAllReduce<SumOp, float, NUM_THREADS>(thread_sum) /
      float(H * W * C_G);

  float thread_sq_sum = 0;
  for (int32_t load_idx = threadIdx.x; load_idx < H / ILP * W * C_G_2;
       load_idx += blockDim.x) {
    auto c_g_2 = load_idx % C_G_2;
    auto w = (load_idx / C_G_2) % W;
    auto h_ilp = ((load_idx / C_G_2) / W);

#pragma unroll ILP
    for (auto ii = 0; ii < ILP; ++ii) {
      half2 valh =
          svals[(h_ilp * ILP + ii) * W * C_G_stride + w * C_G_stride + c_g_2];
      float2 val = __half22float2(valh);
      thread_sq_sum += (val.x - block_mean) * (val.x - block_mean) +
          (val.y - block_mean) * (val.y - block_mean);
    }
  }
  // PyTorch uses biased estimate of std-dev.
  const float block_inv_std = __frsqrt_rn(
      BlockAllReduce<SumOp, float, NUM_THREADS>(thread_sq_sum) /
          float(H * W * C_G) +
      epsilon);

  for (int32_t load_idx = threadIdx.x; load_idx < H / ILP * W * C_G_2;
       load_idx += blockDim.x) {
    auto c_g_2 = load_idx % C_G_2;
    auto w = (load_idx / C_G_2) % W;
    auto h_ilp = ((load_idx / C_G_2) / W);

    auto g = __half22float2(
        *reinterpret_cast<const __half2*>(&gamma[start_c + c_g_2 * 2]));
    g.x *= block_inv_std;
    g.y *= block_inv_std;
    auto b = __half22float2(
        *reinterpret_cast<const __half2*>(&beta[start_c + c_g_2 * 2]));

#pragma unroll ILP
    for (auto ii = 0; ii < ILP; ++ii) {
      __half2* src =
          &svals[(h_ilp * ILP + ii) * W * C_G_stride + w * C_G_stride + c_g_2];
      __half2* dst = reinterpret_cast<__half2*>(
          &(Y[n * strides[0] + (h_ilp * ILP + ii) * strides[1] +
              w * strides[2] + (start_c + c_g_2 * 2)]));

      auto fsrc = __half22float2(*src);
      float2 result;
      result.x = (fsrc.x - block_mean) * g.x + b.x;
      result.y = (fsrc.y - block_mean) * g.y + b.y;
      if (FuseSwish) {
        result.x = result.x * sigmoid(result.x);
        result.y = result.y * sigmoid(result.y);
      }
      *dst = __float22half2_rn(result);
    }
  }
}

template <bool FuseSwish, int H, int W, int C, int num_groups>
hipError_t invokeWelfordGroupNorm(
    half* output,
    half* input,
    half* gamma,
    half* beta,
    int N,
    const float eps,
    hipStream_t stream) {
  int max_vec_size = 8;
  while ((C / num_groups) % max_vec_size != 0) {
    max_vec_size /= 2;
  }

  constexpr int64_t block_size = 1024;
  // counts w.r.t. type half
  const int64_t elems_per_group_channel = C / num_groups;
  const int64_t elems_per_block = (H * W * C) / num_groups;
  const int64_t batch_stride = H * W * C;
  const int64_t group_stride = elems_per_group_channel;

  CHECK_EQ(elems_per_group_channel % max_vec_size, 0);
  CHECK_EQ(batch_stride % max_vec_size, 0);
  CHECK_EQ(group_stride % max_vec_size, 0);
  const int64_t v_elems_per_group_channel =
      elems_per_group_channel / max_vec_size;
  const int64_t v_elems_per_block = elems_per_block / max_vec_size;
  const int64_t v_batch_stride = batch_stride / max_vec_size;
  const int64_t v_group_stride = group_stride / max_vec_size;
  const int64_t v_num_rows = v_elems_per_block / v_elems_per_group_channel;
  const int64_t v_row_stride = C / max_vec_size;

  dim3 grid(N, num_groups);

#define __HANDLE_ONE_VEC(vec_type, vec_size)           \
  case vec_size: {                                     \
    groupnorm_welford_fp16<vec_type, float, FuseSwish> \
        <<<grid, block_size, 0, stream>>>(             \
            reinterpret_cast<vec_type*>(output),       \
            reinterpret_cast<vec_type*>(input),        \
            reinterpret_cast<vec_type*>(gamma),        \
            reinterpret_cast<vec_type*>(beta),         \
            eps,                                       \
            v_elems_per_block,                         \
            v_elems_per_group_channel,                 \
            v_batch_stride,                            \
            v_group_stride,                            \
            v_num_rows,                                \
            v_row_stride);                             \
    GROUP_NORM_CUDA_CHECK_LAUNCH();                    \
    break;                                             \
  }

  switch (max_vec_size) {
    __HANDLE_ONE_VEC(uint4, 8)
    __HANDLE_ONE_VEC(uint2, 4)
    __HANDLE_ONE_VEC(unsigned, 2)
    __HANDLE_ONE_VEC(half, 1)
    default:
      throw std::runtime_error("Invalid max_vec_size\n");
  }

#undef __HANDLE_ONE_VEC
  return hipSuccess;
}

template <bool FuseSwish, int H, int W, int C, int G>
hipError_t invokeGroupNorm(
    half* output,
    half* input,
    half* gamma,
    half* beta,
    int N,
    const float eps,
    const int max_smem_size,
    hipStream_t stream) {
  constexpr auto C_G = C / G;
  constexpr auto C_G_2 = C_G / 2;
  constexpr int ILP = 8;

  // Use a little big more shared_memory to reduce occupancy and boost perf.
  constexpr int MEM_BANK_CONFLICT = 1;

  // Bank conflict doesn't seem to matter to perf
  constexpr int BANK_CONFLICT = 0;

  const auto smem = H * W * (C_G_2 + MEM_BANK_CONFLICT) * 2 * sizeof(uint16_t);

  // C_G must be even, or we can have misaligned address for cp.async
  // reserve some shared_mem for block reduction
  if (H % 8 == 0 && C_G % 2 == 0 && smem <= max_smem_size - 1000) {
    GROUP_NORM_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
        group_norm_smem<FuseSwish), H, W, C, C_G, ILP, BANK_CONFLICT>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem));

    constexpr int num_threads = std::min(1024, H / ILP * W * C_G_2);

    dim3 block(num_threads);
    group_norm_smem<FuseSwish, H, W, C, C_G, ILP, BANK_CONFLICT, num_threads>
        <<<dim3(G, N), block, smem, stream>>>(
            input, output, gamma, beta, N, eps);
  } else {
    return invokeWelfordGroupNorm<FuseSwish, H, W, C, G>(
        output, input, gamma, beta, N, eps, stream);
  }

  // GROUP_NORM_CUDA_CHECK_LAUNCH();
  // TODO: last error is 0, but invoked error logging no error
  return hipGetLastError();
}

#endif /* GROUPNORM_KERNEL_CUH */


}  // namespace


hipError_t groupnorm_swish_280(half* output,
                          half* input,
                          half* gamma,
                          half* beta,
                          int N,
                          const float eps,
                          const int max_smem_size,
                          hipStream_t stream)
    
{
    return invokeGroupNorm<true, 16, 16, 640, 32>(
            output,
            input,
            gamma,
            beta,
            N,
            eps,
            max_smem_size,
            stream);
}
    