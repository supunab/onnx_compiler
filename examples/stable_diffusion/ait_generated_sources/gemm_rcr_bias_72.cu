
#include <iostream>
#include <memory>
#include <random>
#include <vector>
#include <iostream>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/epilogue/thread/linear_combination_silu.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/device_memory.h"



#define CUTLASS_CHECK(status)                                                         \
  {                                                                                   \
    cutlass::Status error = status;                                                   \
    if (error != cutlass::Status::kSuccess) {                                         \
      auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +              \
          cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);         \
      std::cerr << msg << std::endl;                                                  \
      throw std::runtime_error(msg);                                                  \
    }                                                                                 \
  }



  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using f769701041f4627d554c265e4398ecd173393509c = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8;

void gemm_rcr_bias_72 (
    cutlass::half_t* a_ptr,
    cutlass::half_t* b_ptr,
    cutlass::half_t* bias_ptr,
    cutlass::half_t* c_ptr,
    uint8_t* workspace,
    int split_k,
    int64_t* a_dim0,
    int64_t* a_dim1,
    int64_t* a_dim2,
    int64_t* b_dim0,
    int64_t* b_dim1,
    int64_t* c_dim0,
    int64_t* c_dim1,
    int64_t* c_dim2,
  hipStream_t stream
  ) {
  
 int64_t M = (*a_dim0) * (*a_dim1);

 int64_t N = (*b_dim0);

 int64_t K = (*a_dim2);
  
  int64_t input_a_batch_stride = M * K;
  int64_t input_a_stride = K;
  int64_t input_a_offset = 0; // default to 0
  int64_t input_b_batch_stride = N * K;
  int64_t input_b_stride = K;
  int64_t input_b_offset = 0; // default to 0
    
  
  
  int64_t output_stride = N;
  int64_t output_offset = 0;
  
    
  
  
  int64_t a_size = 1;

    a_size *= *a_dim0;

    a_size *= *a_dim1;

    a_size *= *a_dim2;

  if (a_size != 0 && !a_ptr) {
    throw std::runtime_error("input a is null!");
  }

  int64_t b_size = 1;

    b_size *= *b_dim0;

    b_size *= *b_dim1;

  if (b_size != 0 && !b_ptr) {
    throw std::runtime_error("input b is null!");
  }

  int64_t c_size = 1;

    c_size *= *c_dim0;

    c_size *= *c_dim1;

    c_size *= *c_dim2;

  if (c_size != 0) {
    if (!c_ptr) {
      throw std::runtime_error("input c is null!");
    }
  } else {
    // output is empty and safe to return
    return;
  }

  // One of the input tensor are empty
  if (a_size == 0 || b_size == 0) {
    return;
  }

  if (!bias_ptr) {
    throw std::runtime_error("bias_ptr is null!");
  }

  
  if (M == 8192 && N == 320 && K == 1280) {
    
//  TODO: cast to right dtype
    using ElementComputeEpilogue = typename f769701041f4627d554c265e4398ecd173393509c::ElementAccumulator;

    typename f769701041f4627d554c265e4398ecd173393509c::Arguments arguments{


    cutlass::gemm::GemmUniversalMode::kGemm,
    {M, N, K},
    split_k,
    {ElementComputeEpilogue(1), ElementComputeEpilogue(1)},
    (void*) (a_ptr + input_a_offset),
    (void*) (b_ptr + input_b_offset),
    (void*) bias_ptr,
    (void*) (c_ptr + output_offset),
    input_a_batch_stride,
    input_b_batch_stride,
    /*bias_batch_stride*/ N,
    /*output_batch_stride*/ M * N,
    input_a_stride,
    input_b_stride,
    /*bias_stride*/ 0,
    output_stride

    };
    f769701041f4627d554c265e4398ecd173393509c gemm_op;

    auto status = gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);
    status = gemm_op.initialize(arguments, workspace, stream);
    CUTLASS_CHECK(status);
    status = gemm_op(stream);
    CUTLASS_CHECK(status);
    return;
  }
      std::cout << "input_ndims0: " << *a_dim0 << std::endl;
      std::cout << "input_ndims1: " << *a_dim1 << std::endl;
      std::cout << "input_ndims2: " << *a_dim2 << std::endl;
      std::cout << "weight_ndims0: " << *b_dim0 << std::endl;
      std::cout << "weight_ndims1: " << *b_dim1 << std::endl;
      std::cout << "output_ndims0: " << *c_dim0 << std::endl;
      std::cout << "output_ndims1: " << *c_dim1 << std::endl;
      std::cout << "output_ndims2: " << *c_dim2 << std::endl;
  throw std::runtime_error(
      "Unsupported workload for this gemm_rcr_bias_72 specialization."
  );
}