
#include <iostream>
#include <string>
#include <stdexcept>
#include "cutlass/cutlass.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include <cutlass/epilogue/thread/linear_combination_bias_relu.h>
#include <cutlass/epilogue/thread/linear_combination_hardswish.h>



#define CUTLASS_CHECK(status)                                                         \
  {                                                                                   \
    cutlass::Status error = status;                                                   \
    if (error != cutlass::Status::kSuccess) {                                         \
      auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +              \
          cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);         \
      std::cerr << msg << std::endl;                                                  \
      throw std::runtime_error(msg);                                                  \
    }                                                                                 \
  }



  // Conv2dFprop Optimized kernel instance "cutlass_tensorop_f16_s16816fprop_optimized_f16_128x256_32x3_nhwc_align8"
  using cutlass_tensorop_f16_s16816fprop_optimized_f16_128x256_32x3_nhwc_align8_base = 
  typename cutlass::conv::kernel::DefaultConv2dFprop<
    cutlass::half_t, 
    cutlass::layout::TensorNHWC,
    cutlass::half_t, 
    cutlass::layout::TensorNHWC,
    cutlass::half_t, 
    cutlass::layout::TensorNHWC,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32 >,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>, // cutlass::gemm::threadblock::GemmSplitKIdentityThreadblockSwizzle<>,
    3,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    8,
    8
  >::Kernel;

using f5d463ef14775fd04f71faa82a7b6b09f4a1f0b06 = cutlass::conv::device::ImplicitGemmConvolution<cutlass_tensorop_f16_s16816fprop_optimized_f16_128x256_32x3_nhwc_align8_base>;



void conv2d_bias_551 (
    cutlass::half_t* in_ptr,
    cutlass::half_t* weight_ptr,
    cutlass::half_t* out_ptr,
    cutlass::half_t* bias_ptr,
    uint8_t* workspace,
    int64_t* batch,
    int64_t* out_ch,
    int64_t* in_ch,
    int64_t* kernel_h,
    int64_t* kernel_w,
    int64_t* in_h,
    int64_t* in_w,
    int64_t* out_batch,
    int64_t* out_h,
    int64_t* out_w,
    int stride,
    int dilation,
    int pad,
    hipStream_t stream
  ) {

  
  int64_t NI = *batch;
  int64_t HI = *in_h;
  int64_t WI = *in_w;
  int64_t CI = *in_ch;
  int64_t CO = *out_ch;
  int64_t KH = *kernel_h;
  int64_t KW = *kernel_w;
  int64_t SH = stride;
  int64_t SW = stride;
  int64_t DH = dilation;
  int64_t DW = dilation;
  int64_t PH = pad;
  int64_t PW = pad;
  int64_t KHEff = (KH - 1) * DH + 1;
  int64_t KWEff = (KW - 1) * DW + 1;
  int64_t NO = NI;
  int64_t HO = (HI + PH + PH - KHEff) / SH + 1;
  int64_t WO = (WI + PW + PW - KWEff) / SW + 1;
  *out_batch = NO;
  *out_h = HO;
  *out_w = WO;
  *out_ch = CO;
  int i32_batch = *batch;
  int i32_in_h = *in_h;
  int i32_in_w = *in_w;
  int i32_in_ch = *in_ch;
  int i32_out_ch = *out_ch;
  int i32_kernel_h = *kernel_h;
  int i32_kernel_w = *kernel_w;
  int i32_out_batch = *out_batch;
  int i32_out_h = *out_h;
  int i32_out_w = *out_w;

  using cutlass::layout::TensorNHWC;
  TensorNHWC layout_A(TensorNHWC::packed(cutlass::make_Coord(i32_batch, i32_in_h, i32_in_w, i32_in_ch)));
  TensorNHWC layout_B(TensorNHWC::packed(cutlass::make_Coord(i32_out_ch, i32_kernel_h, i32_kernel_w, i32_in_ch)));
  TensorNHWC layout_C(TensorNHWC::packed(cutlass::make_Coord(i32_out_batch, i32_out_h, i32_out_w, i32_out_ch)));

  cutlass::conv::Conv2dProblemSize problem_size(
    {i32_batch, i32_in_h, i32_in_w, i32_in_ch},
        {i32_out_ch, i32_kernel_h, i32_kernel_w, i32_in_ch},
        {pad, pad, pad, pad},
        {stride, stride},
        {dilation, dilation},
        {i32_out_batch, i32_out_h, i32_out_w, i32_out_ch},
        cutlass::conv::Mode::kCrossCorrelation,
        1
  );

  
  if (NI == 2 && HI == 16 && WI == 16 && CI == 2560) {
    
    using ElementComputeEpilogue = typename f5d463ef14775fd04f71faa82a7b6b09f4a1f0b06::ElementCompute;
//  TODO: cast to right dtype
    typename f5d463ef14775fd04f71faa82a7b6b09f4a1f0b06::Arguments arguments{
        problem_size,
        {(cutlass::half_t*)(in_ptr), layout_A},
        {(cutlass::half_t*)(weight_ptr), layout_B},
        {(cutlass::half_t*)(bias_ptr), cutlass::layout::TensorNHWC::Stride(0)},
        {(cutlass::half_t*)(out_ptr), layout_C},
        {ElementComputeEpilogue(1), ElementComputeEpilogue(1)},
    };
    f5d463ef14775fd04f71faa82a7b6b09f4a1f0b06 implicit_gemm_op;

    auto status = implicit_gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);
    status = implicit_gemm_op.initialize(arguments, workspace);
    CUTLASS_CHECK(status);
    status = implicit_gemm_op(stream);
    CUTLASS_CHECK(status);
    return;
  }
  throw std::runtime_error(
      "Unsupported workload for this conv2d specialization."
  );
}